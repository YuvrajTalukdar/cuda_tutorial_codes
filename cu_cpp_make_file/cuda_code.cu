
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void explain_threads()
{
    printf("\nthreadId x: %d, y: %d, z: %d",threadIdx.x,threadIdx.y,threadIdx.z);
}

__global__ void explain_blocks()
{
    printf("\nblockId x: %d, y: %d, z: %d",blockIdx.x,blockIdx.y,blockIdx.z);
}

void start_kernels()
{
    dim3 thread_set(2,2,2);//no of instance run will be 2*2*2 here.threadIdx and blockIdx has 3 dimentions x,y,z.
    explain_threads<<<1,thread_set>>>();//<<<block_idx,thread_idx>>>
    explain_blocks<<<thread_set,1>>>();//<<<block_idx,thread_idx>>>

    hipDeviceSynchronize();
}