
#include <hip/hip_runtime.h>
#include<iostream>
#include<unistd.h>
#include<vector>

using namespace std;

struct data_struct
{
    int id;
    float number;
    string name;
};

__global__ void print_data(data_struct *data)
{
    printf("ID: %d",data->id);
    printf("\nnumber: %f",data->number);
    //printf("\nname: %s",data->name);//std::string not supported in cuda functions
}

int main()
{
    //alocating structure obj
    data_struct d1;
    d1.id=145;
    d1.name="Yuvraj Talukdar";
    d1.number=478.365;
    data_struct *d1_g;
    hipMalloc((void**)&d1_g,sizeof(data_struct));
    hipMemcpy(d1_g,&d1,sizeof(data_struct),hipMemcpyHostToDevice);
    print_data<<<1,1>>>(d1_g);

    sleep(3);
    return 0;
}