#include "hip/hip_runtime.h"
#include<thrust/device_vector.h>
//#include<thrust/host_vector.h>

#include<vector>
#include<cstdio>
#include<iostream>

using namespace std;

struct data_struct2
{
    int id2;
    float number2;
    double number_double;
};

struct data_struct
{
    int id;
    float number;
    data_struct2 num_arr[5];//arrays inside vector allowed
    //vector<data_struct2> data_vec2;//vectors inside vectors not allowed as far as i know in cuda.
    //string name;//presence of string data used inside the kernel will result in compilation error
};

__global__ void kernel (data_struct* pd_vec)
{
    printf("blockIdx: %d id: %d number: %f\n", blockIdx.x, pd_vec[blockIdx.x].id,pd_vec[blockIdx.x].number);
    data_struct2* ds2=pd_vec[blockIdx.x].num_arr;
    printf("    threadIdx: %d id: %d number: %f number_double: %f\n",threadIdx.x,(ds2+threadIdx.x)->id2,(ds2+threadIdx.x)->number2,(ds2+threadIdx.x)->number_double);
}

int main()
{
    vector<data_struct> data_vector;
    for(int a=0;a<5;a++)
    {
        data_struct d1;
        d1.id=a;
        d1.number=8.965*(a+1);
        for(int b=0;b<5;b++)
        {
            d1.num_arr[b].id2=b;
            d1.num_arr[b].number2=b*10;
            d1.num_arr[b].number_double=b*45.3;
        }
        data_vector.push_back(d1);
    }
    

    thrust::device_vector<data_struct> d_vec = data_vector;
    data_struct* pd_vec = thrust::raw_pointer_cast(d_vec.data());

    int n = data_vector.size();
    int size_of_inner_array=5;
    kernel<<<size_of_inner_array, n>>>(pd_vec);//<<<block_idx,thread_idx>>>

    return 0;
}