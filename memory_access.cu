
#include <hip/hip_runtime.h>
#include<iostream>

using namespace std;

__global__ void thread_memory_analysis()
{
    int auto_var=1;//auro var. Stored in registers, and if size grows moved to local memory, accessible only by current thread
    __shared__ int shared_var;//shared var. Stored in shared memory, accessible from all threads in its block.
    if(threadIdx.x==0)
    {   
        auto_var=auto_var+5;
        shared_var=5;
        printf("\nauto_var: %d shared_var: %d threadIdx: %d",auto_var,shared_var,threadIdx.x);
    }
    else
    {   
        shared_var+=threadIdx.x;
        printf("\nauto_var: %d shared_var: %d threadIdx: %d",auto_var,shared_var,threadIdx.x);
    }
}
//it looks like data to be allocated to gpu vram, we need to use traditional memory allocation and data copy.
__global__ void block_memory_analysis()
{
    int auto_var=1;//auto var. Stored in registers, accessible only by current thread
    __shared__ int shared_var;//shared var. Stored in shared memory, accessible from all threads in its block.
    if(blockIdx.x==0)
    {   
        auto_var=auto_var+8;
        printf("\nauto_var: %d shared_var: %d blockIdx: %d",auto_var,shared_var,blockIdx.x);
    }
    else
    {   shared_var+=blockIdx.x;
        printf("\nauto_var: %d shared_var: %d blockIdx: %d",auto_var,shared_var,blockIdx.x);}
} 

int main()
{
    thread_memory_analysis<<<1,10>>>();//<<<block_idx,thread_idx>>>
    block_memory_analysis<<<10,1>>>();
    hipDeviceSynchronize();
    return 0;
}