#include "hip/hip_runtime.h"
#include<thrust/device_vector.h>
//#include<thrust/host_vector.h>

#include<vector>
#include<cstdio>
#include<iostream>
#include<unistd.h>

using namespace std;

__global__ void explain_threads()
{
    printf("\nthreadId x: %d, y: %d, z: %d",threadIdx.x,threadIdx.y,threadIdx.z);
}

__global__ void explain_blocks()
{
    printf("\nblockId x: %d, y: %d, z: %d",blockIdx.x,blockIdx.y,blockIdx.z);
}

int main()
{
    dim3 thread_set(2,2,2);//no of instance run will be 2*2*2 here.threadIdx and blockIdx has 3 dimentions x,y,z.
    explain_threads<<<1,thread_set>>>();//<<<block_idx,thread_idx>>>
    explain_blocks<<<thread_set,1>>>();//<<<block_idx,thread_idx>>>
    //hipDeviceSynchronize();// deprecated version of hipDeviceSynchronize
    hipDeviceSynchronize();//halts execution in the CPU/host thread (that the hipDeviceSynchronize was issued in) until the GPU has finished processing all previously requested cuda tasks (kernels, data copies, etc.)
    //hipStreamSynchronize();//waits for the completion of onlt the streames provided as the parameters, for anything else done in gpu it do not wait
    sleep(1);

    return 0;
}