#include "hip/hip_runtime.h"
#include<thrust/device_vector.h>
#include<vector>
#include<cstdio>
#include<iostream>

using namespace std;

struct data_struct
{
    int id;
    float number;
    //string name;//presence of string data used inside the kernel will result in compilation error
};

__global__ void kernel (data_struct* pd_vec, int n)
{
    if(threadIdx.x < n)
    {   printf("threadIdx: %d id: %d number: %f\n", threadIdx.x, pd_vec[threadIdx.x].id,pd_vec[threadIdx.x].number);}
}

int main()
{
    vector<data_struct> data_vector;
    for(int a=0;a<10;a++)
    {
        data_struct d1;
        d1.id=a;
        d1.number=8.965*(a+1);
        data_vector.push_back(d1);
    }
    //{

    thrust::device_vector<data_struct> d_vec = data_vector;
    data_struct* pd_vec = thrust::raw_pointer_cast(d_vec.data());

    int n = data_vector.size();
    kernel<<<1, n>>>(pd_vec, n);//<<<block_idx,thread_idx>>>
    //hipDeviceSynchronize();//Wait for compute device to finish. If kernel calls are considered as thread calls, than hipDeviceSynchronize is like the join function.
    //}
    //hipDeviceReset();//Destroy all allocations and reset all state on the current device in the current process.

    return 0;
}