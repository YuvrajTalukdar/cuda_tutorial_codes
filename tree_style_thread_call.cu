//use -arch=sm_86 or sm_80 for ampear architecture
//-rdc=true is required here

#include <hip/hip_runtime.h>
#include<cstdio>
#include<iostream>
#include<unistd.h>

using namespace std;

int host_fx()//__host__ is called only from host
{
    printf("\nCalled from host fx\n");
    return 5;
}

__device__ void fx2()
{
    printf("\ncall from fx2");
}

__device__ void fx()//__device__ this acts as a normal function called from the kernel
{
    printf("\nfunction1 x: %d, y: %d, z: %d",threadIdx.x,threadIdx.y,threadIdx.z);
    fx2();
}

__global__ void kernel2(bool kernel_call)//__global__  this is the kernel function, only kernel functions can be called in multi threaded mode
{
    if(!kernel_call)
    {   printf("\nkernel2 x: %d, y: %d, z: %d",threadIdx.x,threadIdx.y,threadIdx.z);}
    else
    {   printf("\nkernel2 x: %d, y: %d, z: %d",threadIdx.x,threadIdx.y,threadIdx.z);}
}

__global__ void kernel1(bool kernel_call)
{
    printf("\nkernel1 x: %d, y: %d, z: %d",threadIdx.x,threadIdx.y,threadIdx.z);
    dim3 thread_group(1,1,2);
    if(!kernel_call)
    {   fx();}
    else
    {   kernel2<<<1,thread_group>>>(kernel_call);}
}

int main()
{
    dim3 thread_set(8,1,1);//no of instance run will be 2*2*2 here
    kernel1<<<1,thread_set>>>(false);//<<<block_idx,thread_idx>>>
    kernel1<<<1,2>>>(true);//<<<block_idx,thread_idx>>>
    hipDeviceSynchronize();//halts execution in the CPU/host thread (that the cudaDeviceSynchronize was issued in) until the GPU has finished processing all previously requested cuda tasks (kernels, data copies, etc.)
    cout<<"\nr= "<<host_fx();
    //cudaStreamSynchronize();//waits for the completion of onlt the streames provided as the parameters, for anything else done in gpu it do not wait
    //sleep(1);

    return 0;
}