/*
This program gets the following details:-
1. No of gpu
2. GPU Name
3. No of blocks
4. No of threads per blocks
5. Total grobal vram
6. Total Shared Memory
*/

#include <hip/hip_runtime.h>
#include<iostream>

using namespace std;

int main()
{
    int nDevices;
    hipGetDeviceCount(&nDevices);
    cout<<"Number of devices: "<<nDevices;
    for(int a=0;a<nDevices;a++)
    {
        cout<<"\nDevice ID: "<<a;
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, a);
        cout<<"\nDevice Name: "<<prop.name;
        cout<<"\nNo of blocks: "<<prop.maxBlocksPerMultiProcessor;
        cout<<"\nNo of thread per Blocks: "<<prop.maxThreadsPerBlock;
        cout<<"\nTotal Global Memory in MB: "<<(float)prop.totalGlobalMem/((float)1024*1024);//vram 
        cout<<"\nTotal Shared Memory in KB: "<<(float)(prop.sharedMemPerBlock)/1024.0;//accessible by the block which owns it. Its like the personal cache for the block.
    }

    return 0;
}