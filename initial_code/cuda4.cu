#include "hip/hip_runtime.h"
//testing async compute
#include<stdio.h>
#include<unistd.h>
#include<iostream>
#include<chrono>
#include<random>
#include"hip/hip_runtime_api.h"
#include"hip/hip_runtime.h"
#include"hip/hip_runtime.h"

using namespace std;
using namespace chrono;

__global__ void infinite_loop()
{
    int x=0;
    while(true)
    {
        x++;x--;
    }
}

int getSPcores(hipDeviceProp_t devProp)
{  
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch (devProp.major){
     case 2: // Fermi
      if (devProp.minor == 1) cores = mp * 48;
      else cores = mp * 32;
      break;
     case 3: // Kepler
      cores = mp * 192;
      break;
     case 5: // Maxwell
      cores = mp * 128;
      break;
     case 6: // Pascal
      if ((devProp.minor == 1) || (devProp.minor == 2)) cores = mp * 128;
      else if (devProp.minor == 0) cores = mp * 64;
      else printf("Unknown device type\n");
      break;
     case 7: // Volta and Turing
      if ((devProp.minor == 0) || (devProp.minor == 5)) cores = mp * 64;
      else printf("Unknown device type\n");
      break;
     case 8: // Ampere
      if (devProp.minor == 0) cores = mp * 64;
      else if (devProp.minor == 6) cores = mp * 128;
      else printf("Unknown device type\n");
      break;
     default:
      printf("Unknown device type\n"); 
      break;
      }
    return cores;
}

void print_device_props() 
{
    int nDevices;
    hipGetDeviceCount(&nDevices);
    printf("Number of devices: %d\n", nDevices);
    for (int i = 0; i < nDevices; i++) 
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (MHz): %d\n",
                prop.memoryClockRate/1024);
        printf("  Memory Bus Width (bits): %d\n",
                prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %.1f\n",
                2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        printf("  Total global memory (Gbytes) %.1f\n",(float)(prop.totalGlobalMem)/1024.0/1024.0/1024.0);
        printf("  Shared memory per block (Kbytes) %.1f\n",(float)(prop.sharedMemPerBlock)/1024.0);
        printf("  minor-major: %d-%d\n", prop.minor, prop.major);
        printf("  Warp-size: %d\n", prop.warpSize);
        printf("  Concurrent kernels: %s\n", prop.concurrentKernels ? "yes" : "no");
        printf("  Concurrent computation/communication: %s\n\n",prop.deviceOverlap ? "yes" : "no");
    }
}

int main()
{
    //infinite_loop<<<10,5>>>();
    int deviceID;
    hipDeviceProp_t props;
    hipGetDevice(&deviceID);
    hipGetDeviceProperties(&props,deviceID);
    int CUDACores = getSPcores(props);
    cout<<"Cores: "<<CUDACores;
    cout<<"\nMultiprocessor Count: "<<props.multiProcessorCount;
    cout<<"\nAsync Engine Count: "<<props.asyncEngineCount;
    print_device_props();
    return 0;
}