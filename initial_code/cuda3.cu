//adding 2 arrays in cuda

#include <hip/hip_runtime.h>
#include<stdio.h>
#include<unistd.h>
#include<iostream>
#include<chrono>
#include<random>

using namespace std;
using namespace chrono;

__global__ void add_by_gpu(int *x,int *y,int *result)
{
    result[blockIdx.x]=x[blockIdx.x]+y[blockIdx.x];
}

__global__ void hello()
{
    printf("\nhello world from gpu\n");
}

int get_random_number(int min,int max)
{
    random_device dev;
    mt19937 rng(dev());
    uniform_int_distribution<std::mt19937::result_type> dist6(min,max); // distribution in range [1, 6]
    return dist6(rng);
}

int main() 
{
    auto point0_1 = high_resolution_clock::now();
    hello<<<1,1>>>();
    auto point0_2 = high_resolution_clock::now();
    auto duration0 = duration_cast<microseconds>(point0_2 - point0_1);
    cout<<"Time required to print hello world by gpu: "<<to_string(duration0.count()/pow(10,6))<<endl;

    auto point1 = high_resolution_clock::now();
    int size=10000;
    int data_size=sizeof(int)*size;
    int x[size],y[size],*result=(int*)malloc(data_size);
    int *x_d,*y_d,*result_d;
    for(int a=0;a<size;a++)
    {
        x[a]=get_random_number(1,size);
        y[a]=get_random_number(1,size);
    }
    auto point2 = high_resolution_clock::now();
    auto duration1 = duration_cast<microseconds>(point2 - point1);
    cout<<"Time required to initialize the arrays(cpu): "<<to_string(duration1.count()/pow(10,6));
    
    auto point3 = high_resolution_clock::now();
    hipMalloc((void **)&x_d,data_size);
    auto point3_2 = high_resolution_clock::now();
    hipMalloc((void **)&y_d,data_size);
    auto point3_3 = high_resolution_clock::now();
    hipMalloc((void **)&result_d,data_size);
    auto point4 = high_resolution_clock::now();
    auto duration2 = duration_cast<microseconds>(point4 - point3);
    cout<<"\nTime required to allocate "<<data_size<<"*3 in gpu : "<<to_string(duration2.count()/pow(10,6));
    auto duration2_2 = duration_cast<microseconds>(point3_2 - point3);
    cout<<"\nTime required to allocate "<<data_size<<"*1 in gpu : "<<to_string(duration2_2.count()/pow(10,6));
    auto duration2_3 = duration_cast<microseconds>(point3_3 - point3_2);
    cout<<"\nTime required to allocate "<<data_size<<"*1 in gpu : "<<to_string(duration2_3.count()/pow(10,6));
    auto duration2_4 = duration_cast<microseconds>(point4 - point3_3);
    cout<<"\nTime required to allocate "<<data_size<<"*1 in gpu : "<<to_string(duration2_4.count()/pow(10,6));

    auto point5 = high_resolution_clock::now();
    hipMemcpy(x_d,&x,data_size,hipMemcpyHostToDevice);
    auto point5_2 = high_resolution_clock::now();
    hipMemcpy(y_d,&y,data_size,hipMemcpyHostToDevice);
    auto point6 = high_resolution_clock::now();
    auto duration3 = duration_cast<microseconds>(point6-point5);
    cout<<"\nTime required to copy "<<data_size<<"*2 to gpu : "<<to_string(duration3.count()/pow(10,6));
    auto duration3_2 = duration_cast<microseconds>(point5_2 - point5);
    cout<<"\nTime required to copy "<<data_size<<"*1 to gpu : "<<to_string(duration3_2.count()/pow(10,6));
    auto duration3_3 = duration_cast<microseconds>(point6 - point5_2);
    cout<<"\nTime required to copy "<<data_size<<"*1 to gpu : "<<to_string(duration3_3.count()/pow(10,6));

    auto point7 = high_resolution_clock::now();
    add_by_gpu<<<size,1>>>(x_d,y_d,result_d);
    auto point8 = high_resolution_clock::now();
    auto duration4 = duration_cast<microseconds>(point8-point7);
    cout<<"\nTime required to complete the addition by gpu : "<<to_string(duration4.count()/pow(10,6));

    auto point9 = high_resolution_clock::now();
    hipMemcpy(result,result_d,data_size,hipMemcpyDeviceToHost);
    hipFree(x_d);
	hipFree(y_d);
    hipFree(result_d);
    auto point10 = high_resolution_clock::now();
    auto duration5 = duration_cast<microseconds>(point10-point9);
    cout<<"\nTime required to copy the results to host and free the vram : "<<to_string(duration5.count()/pow(10,6));

    cout<<"\n\n";
    bool ok=true;
    for(int a=0;a<size;a++)
    {
        //cout<<"\nx="<<x[a]<<" y="<<y[a]<<" r="<<*(result+a);
        if(x[a]+y[a]!=*(result+a))
        {   ok=false;break;}
    } 
    if(ok)
    {   cout<<"\nAddition done correctly.";}
    else
    {   cout<<"\nAddition could not be done correctly.";}
    //sleep(5);
    return 0;
}
